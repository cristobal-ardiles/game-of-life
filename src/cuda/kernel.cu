#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "stdio.h"
__device__ void check_neighbour(int i, int j, int rel_i, int rel_j, int n, int m, unsigned char* state, int& counter){
  int neigh_i = (i+rel_i) % n; 
  int neigh_j = (j+rel_j) % m; 
  int neigh_index = neigh_i*m + neigh_j; 
  if (state[neigh_index]) counter++; 
}

__global__ void game_of_cuda(unsigned char *curr, unsigned char *next, 
                            int n, int m){
  // Simulates a single step on game of life. 
  int size = n*m;
  int idx = blockDim.x * blockIdx.x + threadIdx.x; 
  printf("I'm thread %d\n", idx); 
  for (int thread = idx; thread < size; thread += gridDim.x*blockDim.x){
    // Get row, column index 
    int i = thread / m; 
    int j = thread % m; 
    int alive = 0; 
    //Check each neighbour
    check_neighbour(i,j,-1,-1,n,m,curr,alive); // North West neighbour 
    check_neighbour(i,j,-1, 0,n,m,curr,alive); // N 
    check_neighbour(i,j,-1, 1,n,m,curr,alive); // NE 
    check_neighbour(i,j, 0,-1,n,m,curr,alive); // W  
    check_neighbour(i,j, 0, 1,n,m,curr,alive); // E 
    check_neighbour(i,j, 1,-1,n,m,curr,alive); // SW  
    check_neighbour(i,j, 1, 0,n,m,curr,alive); // S  
    check_neighbour(i,j, 1, 1,n,m,curr,alive); // SE

    // Apply rules. TODO: fix divergence. We have to figure out how to run this concurrently
    if (curr[thread]){
      // If cell is alive
      next[thread] = (alive < 2 || alive >= 4) ? 0 : 1; 
    } else {
      // If cell is dead
      next[thread] = alive == 3 ? 1 : 0;
    }
  }
}

